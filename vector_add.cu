
#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA kernel to add two vectors
__global__ void vectorAdd(int *a, int *b, int *c, int n) {
    // Calculate global thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Bounds check
    if (tid < n) {
        c[tid] = a[tid] + b[tid];
    }
}

// Host function to initialize vectors and call CUDA kernel
extern "C" void cuda_vector_add(int *h_a, int *h_b, int *h_c, int n) {
    int *d_a, *d_b, *d_c; // Device arrays

    // Allocate memory on GPU
    hipMalloc((void **)&d_a, n * sizeof(int));
    hipMalloc((void **)&d_b, n * sizeof(int));
    hipMalloc((void **)&d_c, n * sizeof(int));

    // Copy input vectors from host to GPU memory
    hipMemcpy(d_a, h_a, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, n * sizeof(int), hipMemcpyHostToDevice);

    // Define block size and grid size
    int block_size = 256;
    int grid_size = (n + block_size - 1) / block_size;

    // Launch CUDA kernel
    vectorAdd<<<grid_size, block_size>>>(d_a, d_b, d_c, n);

    // Copy result vector from GPU to host memory
    hipMemcpy(h_c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}
